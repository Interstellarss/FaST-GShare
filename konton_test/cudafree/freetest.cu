#include <iostream>
#include <hip/hip_runtime.h>

#define N 512  // 矩阵的维度

__global__ void matrixAdd(float* A, float* B, float* C, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < width) {
        int index = y * width + x;
        C[index] = A[index] + B[index];
    }
}

int main() {
    // 矩阵大小
    int size = N * N * sizeof(float);
    
    // 主机端数据
    float *h_A, *h_B, *h_C;

    // 设备端数据
    float *d_A, *d_B, *d_C;

    // 分配主机内存
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // 初始化矩阵 A 和 B
    for (int i = 0; i < N * N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // 分配设备内存
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 设定线程和块的数量
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用内核函数进行矩阵加法
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 释放不必要的 GPU 内存（例如不再需要的 d_A 和 d_B）
    hipFree(d_A);
    hipFree(d_B);

    // 从设备复制结果到主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 打印前几个结果以验证
    for (int i = 0; i < 10; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // 释放设备内存
    hipFree(d_C);

    // 释放主机内存
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}